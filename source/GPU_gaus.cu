#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
// Filename : GPU_gaus.cu
// Created  : 04/10/2018
// Author   : Michael Davis
//
// Description : .cu CUDA C code designed to perform GPU accelerated Gaussian Blur
//               on 24bit bitmap RGB data.
//
// -----------------------------------------------------------------------------
#include "GPU_gaus.h"
#include <iostream>


// -----------------------------------------------------------------------------
// Function Name : gpuErrchk, gpuAssert
//
// Description : CUDA call error checking and handling. This C function was copied
//               from stack overflows website from user talonmies comment.
//
// Credit :
// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
// -----------------------------------------------------------------------------
extern "C" {
      #include <stdio.h>
      #define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
      inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
      {
         if (code != hipSuccess)
         {
            fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
            if (abort) exit(code);
         }
      }
}



// -----------------------------------------------------------------------------
// Function Name : cuda2dto1d
//
// Description : Device function to be run and called by the GPU.
//               Helper function to convert 2D coordinates to a 1D index.
//
// Inputs : w - Width of 2D co-ordinate space.
//          x, y - Co-ords to be converted.
// -----------------------------------------------------------------------------
__device__ int cuda2dto1d(int w, int x, int y)
{
      return ((y * w) + x);
}



// -----------------------------------------------------------------------------
// Function Name : cudaIsInBounds
//
// Description : Device function to be run and called by the GPU.
//               Helper function to check if a 2D co-ordinate position is in bounds.
//
// Inputs : width, height - Width and Height of 2D co-ordinate space.
//          x, y - Co-ords to be checked.
// -----------------------------------------------------------------------------
__device__ bool cudaIsInBounds(int width, int height, int x, int y)
{
      return x < width && x >= 0 && y < height && y >=0 ? true : false;
}


// -----------------------------------------------------------------------------
// Function Name : getSharedGlobalCoOrds
//
// Description : retrieve cartesian coordinate values of a 2D shared memory structure.
//
// Inputs : sindx - relative index in shared memory.
//		bpos  - gloabal 2D coordinates of 2D cuda block.
//		sDim  - Dimensions of 2D shared memory.
//		w, h  - width and height of image.
//		hw	- half width of filter (filterwidth - 1) / 2
//
// Output : out - out co-ordinates relative to the image of the shared memory
//		bool - if the coordinates are within the image or not.
//         
// -----------------------------------------------------------------------------

__device__ bool getSharedGlobalCoOrds(int sindx, dim3 bpos, dim3 sDim, int w, int h, int hw, dim3 *out)
{
	dim3 spos(sindx%sDim.x, sindx/sDim.x);
	dim3 gpos(bpos.x+spos.x-hw, bpos.y+spos.y-hw);
	if (cudaIsInBounds(w, h, gpos.x, gpos.y))
	{
		*out = gpos;
		return true;
	}
	return false;
}


// -----------------------------------------------------------------------------
// Function Name : getSharedIndx
//
// Description : retrieve realtive shared memory index based on 2D global coordinates.
//
// Inputs : gpos  - globaal 2D coordinates of 2D position.
//		bpos  - gloabal 2D coordinates of 2D cuda block.
//		sDim  - Dimensions of 2D shared memory.
//		hw	- half width of filter (filterwidth - 1) / 2
//
// Output : indx - out index relative to the image of the shared memory
//		bool - if the coordinates are within the shared memory dimensions.
//         
// -----------------------------------------------------------------------------
__device__ bool getSharedIndx(dim3 gpos, dim3 bpos, dim3 sDim, int hw, int *indx)  
{
	dim3 spos(gpos.x - bpos.x + hw, gpos.y - bpos.y + hw);
	if (cudaIsInBounds(sDim.x, sDim.y, spos.x, spos.y))
	{
		*indx = spos.y * sDim.x + spos.x;
		return true;
	}
	return false;
}


// -----------------------------------------------------------------------------
// Function Name : cudaGaussianBlur
//
// Description : Global CUDA Kernel function that can be called from host code.
//               Uses CUDA acceleration to create gaussian blurred data.
//               Takes in original RGB channel data and uses a convolutional
//               filter of width fil_w to generate weighted blur values for each pixel.
//               Filter must be an odd valued width.
//
// Input : *f_buff, fil_w - Filter buffer data, should be normalized in order to maintain
//                          image "energy", this stops darkening or lightening effects of
//                          the image upon computation.
//
//         *r_buff, *g_buff, *b_buff - RGB image data channels inputs.
//
//         w, h - Width and heght of the image
//
//         *r_out, *g_out, *b_out - RGB image data channels output arrays.
// -----------------------------------------------------------------------------
__global__ void cudaGaussianBlur(float *f_buff,  /* In  - Filter buffer data         */
                                 int fil_w,      /* In  - Fileter width              */
                                 int fil_off,    /* In  - Filter offset value        */
                                 unsigned char *r,         /* In  - Red channel buffer data    */
                                 unsigned char *g,         /* In  - Green channel buffer data  */
                                 unsigned char *b,         /* In  - Blue channel buffer data   */
                                 int w,          /* In  - Image width                */
                                 int h,          /* In  - Image height               */
                                 unsigned char *r_out,     /* Out - Red channel buffer data    */
                                 unsigned char *g_out,     /* Out - Red channel buffer data    */
                                 unsigned char *b_out,      /* Out - Red channel buffer data    */
					   dim3 sharedDim
                                 )
{
      // Iterable variables
      int x, y, f_x, f_y;

      // Temporary float values for each colour channel
      float r_val, g_val, b_val;

      // Sample coordinates and 1D texture indexes
      int x_sam, y_sam;
      int fil_index, shared_index;

      x = blockIdx.x * blockDim.x + threadIdx.x;
      y = blockIdx.y * blockDim.y + threadIdx.y;
	int ind = y * w + x;
	int local_index = threadIdx.y * blockDim.x + threadIdx.x;
	int localArea = blockDim.x * blockDim.y;

	int sharedArea = sharedDim.x * sharedDim.y;
	dim3 bpos(blockIdx.x * blockDim.x, blockIdx.y * blockDim.y);

	extern __shared__ unsigned char shared_mem[];
	unsigned char *s_r = shared_mem;
	unsigned char *s_g = (unsigned char*)&shared_mem[sharedArea];
	unsigned char *s_b = (unsigned char*)&s_g[sharedArea];

	
	int i = 0;
	while ((i * localArea) + (local_index) < sharedArea)
	{
		int sindx = (i * localArea) + local_index;
		dim3 gPos;
		if (getSharedGlobalCoOrds(sindx, bpos, sharedDim, w, h, -fil_off, &gPos))
		{
			s_r[sindx] = r[gPos.y * w + gPos.x];
			s_g[sindx] = g[gPos.y * w + gPos.x];
			s_b[sindx] = b[gPos.y * w + gPos.x];
		}
		i++;
	}

	__syncthreads();

      // Check index is wthin image bounds
      if (cudaIsInBounds(w, h, x, y))
      {
            // Initialize temporary values for colour channels
            r_val = 0.0f;
            g_val = 0.0f;
            b_val = 0.0f;

            // Begin filter convolution.
            for (f_x = 0; f_x < fil_w; f_x++)
            {
                  for (f_y = 0; f_y < fil_w; f_y++)
                  {

                        // Calculate sample coordinates for original image
                        x_sam = x + f_x + fil_off;
                        y_sam = y + f_y + fil_off;

                        // If ample co-ords out of bounds, sample from current pixel.
                        if (cudaIsInBounds(w, h, x_sam, y_sam) == false)
                        {
                              x_sam = x;
                              y_sam = y;
                        }

                        // Caluclate 1D array index values.
                        fil_index = cuda2dto1d(fil_w, f_x, f_y);
				
				if (getSharedIndx(dim3(x_sam, y_sam), bpos, sharedDim, -fil_off, &shared_index))
				{ 

					// Increment temporary pixel colour values.
					r_val += ((float) s_r[shared_index] * (float) f_buff[fil_index]);
					g_val += ((float) s_g[shared_index] * (float) f_buff[fil_index]);
					b_val += ((float) s_b[shared_index] * (float) f_buff[fil_index]);
				}
                  }
            }

            // Assign values to output arrays.
          	r_out[ind] = (unsigned char)(r_val);
          	g_out[ind] = (unsigned char)(g_val);
          	b_out[ind] = (unsigned char)(b_val);
      }
}



// -----------------------------------------------------------------------------
// Function Name : GPUAcceleratedGaussianBlur
//
// Description : Callable GPU acceslerated gaussian blur.
//               Uses CUDA acceleration to create gaussian blurred data.
//               Takes in original RGB channel data and uses a convolutional
//               filter of width fil_w to generate weighted blur values for each
//               pixel. Filter must be an odd valued width.
//
// Input : *f_buff, fil_w - Filter buffer data, should be normalized in order to
//                          maintain image "energy", this stops darkening or
//                          lightening effects of the image upon computation.
//
//         *r_buff, *g_buff, *b_buff - RGB image data channels.
//
//         w, h - Width and heght of the image
// -----------------------------------------------------------------------------
void GPUAcceleratedGaussianBlur(float *f_buff, /* In  - Filter buffer data  */
                                int fil_w,     /* In  - Fileter width       */
                                unsigned char *r_buff,   /* Out - Red channel data    */
                                unsigned char *g_buff,   /* Out - Green channel data  */
                                unsigned char *b_buff,   /* Out - Blue channel data   */
                                int w,         /* In  - Image width         */
                                int h)         /* In  - Image height        */
{

      // ---- Check filter size and generate offset value
      if (fil_w % 2 != 1) return;
      int f_offset = -((fil_w - 1) / 2);

      // ---- Define size variables to allocate space on the GPU ----
      size_t bmp_size = w * h * sizeof(unsigned char);
      size_t fil_size = fil_w * fil_w * sizeof(float);

      // ---- Allocate device memory ----
      unsigned char *d_r, *d_g, *d_b;
      unsigned char *d_r_out, *d_g_out, *d_b_out;
      float *d_f;
      gpuErrchk( hipMalloc(&d_f, fil_size) );

      gpuErrchk( hipMalloc(&d_r, bmp_size) );
      gpuErrchk( hipMalloc(&d_g, bmp_size) );
      gpuErrchk( hipMalloc(&d_b, bmp_size) );

      gpuErrchk( hipMalloc(&d_r_out, bmp_size) );
      gpuErrchk( hipMalloc(&d_g_out, bmp_size) );
      gpuErrchk( hipMalloc(&d_b_out, bmp_size) );

      // ---- Copy data from host memory to device memory ----
      gpuErrchk( hipMemcpy(d_r, r_buff,  bmp_size, hipMemcpyHostToDevice) );
      gpuErrchk( hipMemcpy(d_g, g_buff,  bmp_size, hipMemcpyHostToDevice) );
      gpuErrchk( hipMemcpy(d_b, b_buff,  bmp_size, hipMemcpyHostToDevice) );
      gpuErrchk( hipMemcpy(d_f, f_buff,  fil_size, hipMemcpyHostToDevice) );

      // ---- Determine kernel launch dimensions --- -
	dim3 bDim = dim3(16, 16);
	dim3 gDim = dim3( 1 + (w + bDim.x - 1) / bDim.x, 1 + (h + bDim.y - 1) / bDim.y );
	int shared_size = (bDim.x + fil_w - 1) * (bDim.y + fil_w - 1) * 3 * sizeof(unsigned char);
	std::cout << "shared size : " << shared_size << std::endl;
	dim3 sharedDim = dim3(bDim.x + fil_w - 1, bDim.y + fil_w - 1);

      // --- Launch Kernel ----
      cudaGaussianBlur<<<gDim, bDim, shared_size>>>(d_f, fil_w, f_offset, d_r, d_g, d_b, w, h, d_r_out, d_g_out, d_b_out, sharedDim);

      // ---- Read data back from device to host ----
      gpuErrchk( hipMemcpy(r_buff, d_r_out, bmp_size, hipMemcpyDeviceToHost) );
      gpuErrchk( hipMemcpy(g_buff, d_g_out, bmp_size, hipMemcpyDeviceToHost) );
      gpuErrchk( hipMemcpy(b_buff, d_b_out, bmp_size, hipMemcpyDeviceToHost) );


      // ---- Free up allocated memory ----
      gpuErrchk( hipFree(d_f) );

      gpuErrchk( hipFree(d_r) );
      gpuErrchk( hipFree(d_g) );
      gpuErrchk( hipFree(d_b) );

      gpuErrchk( hipFree(d_r_out) );
      gpuErrchk( hipFree(d_g_out) );
      gpuErrchk( hipFree(d_b_out) );

}
